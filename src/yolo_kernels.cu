#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

extern "C" {
#include "network.h"
#include "detection_layer.h"
#include "cost_layer.h"
#include "utils.h"
#include "parser.h"
#include "box.h"
#include "image.h"
#include <sys/time.h>
}

/* Change class number here */
#define CLS_NUM 6

#ifdef OPENCV
#include "opencv2/highgui/highgui.hpp"
#include "opencv2/imgproc/imgproc.hpp"
extern "C" IplImage* image_to_Ipl(image img, int w, int h, int depth, int c, int step);
extern "C" image ipl_to_image(IplImage* src);
extern "C" void convert_yolo_detections(float *predictions, int classes, int num, int square, int side, int w, int h, float thresh, float **probs, box *boxes, int only_objectness);
extern "C" void draw_yolo(image im, int num, float thresh, box *boxes, float **probs);

extern "C" char *voc_names[];
extern "C" image voc_labels[];

static float **probs;
static box *boxes;
static network net;
static image in   ;
static image in_s ;
static image det  ;
static image det_s;
static image disp ;
static cv::VideoCapture cap;
static cv::VideoWriter cap_out;
static float fps = 0;
static float demo_thresh = 0;
static int w, h, depth, c, step= 0;
static int MODE = -1;

void *fetch_in_thread(void *ptr)
{
    cv::Mat frame_m;
    cap >> frame_m;
    IplImage frame = frame_m;

if(step == 0)
{
    w = frame.width;
    h = frame.height;
    c = frame.nChannels;
    depth= frame.depth; 
    step = frame.widthStep;
}

    in = ipl_to_image(&frame);
    rgbgr_image(in);
    in_s = resize_image(in, net.w, net.h);
    return 0;
}

void *detect_in_thread(void *ptr)
{
    float nms = .4;

    detection_layer l = net.layers[net.n-1];
    float *X = det_s.data;
    float *predictions = network_predict(net, X);
    free_image(det_s);
    convert_yolo_detections(predictions, l.classes, l.n, l.sqrt, l.side, 1, 1, demo_thresh, probs, boxes, 0);
    if (nms > 0) do_nms(boxes, probs, l.side*l.side*l.n, l.classes, nms);
    printf("\033[2J");
    printf("\033[1;1H");
    printf("\nFPS:%.0f\n",fps);
    printf("Objects:\n\n");
    draw_detections(det, l.side*l.side*l.n, demo_thresh, boxes, probs, voc_names, voc_labels, CLS_NUM);

    if(MODE == 1)
    {
        IplImage* outputIpl= image_to_Ipl(det, w, h, depth, c, step);
        cv::Mat outputMat = cv::cvarrToMat(outputIpl, true);
        /*
        cvNamedWindow("image", CV_WINDOW_AUTOSIZE);
        cvShowImage("image", outputIpl); 
        cvWaitKey(1);  
        */
        cvReleaseImage(&outputIpl);
        cap_out << outputMat;
        outputMat.release();
     }

    return 0;
}

extern "C" void demo_yolo(char *cfgfile, char *weightfile, float thresh, int cam_index, char *videofile)
{
    demo_thresh = thresh;
    printf("YOLO demo\n");
    net = parse_network_cfg(cfgfile);
    if(weightfile){
        load_weights(&net, weightfile);
    }
    set_batch_network(&net, 1);

    srand(2222222);

if(cam_index != -1)
{
    MODE = 0; 
    cv::VideoCapture cam(cam_index);
    cap = cam;
    if(!cap.isOpened()) error("Couldn't connect to webcam.\n");
}
else 
{
    MODE = 1;
    printf("Video File name is: %s\n", videofile);
    cv::VideoCapture videoCap(videofile);
    cap = videoCap;
    if(!cap.isOpened()) error("Couldn't read video file.\n");

    cv::Size S = cv::Size((int)videoCap.get(CV_CAP_PROP_FRAME_WIDTH), (int)videoCap.get(CV_CAP_PROP_FRAME_HEIGHT));
    cv::VideoWriter outputVideo("out.avi", CV_FOURCC('D','I','V','X'), videoCap.get(CV_CAP_PROP_FPS), S, true);
    if(!outputVideo.isOpened()) error("Couldn't write video file.\n");
    cap_out = outputVideo;
}
 
    detection_layer l = net.layers[net.n-1];
    int j;

    boxes = (box *)calloc(l.side*l.side*l.n, sizeof(box));
    probs = (float **)calloc(l.side*l.side*l.n, sizeof(float *));
    for(j = 0; j < l.side*l.side*l.n; ++j) probs[j] = (float *)calloc(l.classes, sizeof(float *));

    pthread_t fetch_thread;
    pthread_t detect_thread;

    fetch_in_thread(0);
    det = in;
    det_s = in_s;

    fetch_in_thread(0);
    detect_in_thread(0);
    disp = det;
    det = in;
    det_s = in_s;

    while(1){
        struct timeval tval_before, tval_after, tval_result;
        gettimeofday(&tval_before, NULL);
        if(pthread_create(&fetch_thread, 0, fetch_in_thread, 0)) error("Thread creation failed");
        if(pthread_create(&detect_thread, 0, detect_in_thread, 0)) error("Thread creation failed");
        show_image(disp, "YOLO");
        free_image(disp);
        cvWaitKey(1);
        pthread_join(fetch_thread, 0);
        pthread_join(detect_thread, 0);

        disp  = det;
        det   = in;
        det_s = in_s;

        gettimeofday(&tval_after, NULL);
        timersub(&tval_after, &tval_before, &tval_result);
        float curr = 1000000.f/((long int)tval_result.tv_usec);
        fps = .9*fps + .1*curr;
    }
}
#else
extern "C" void demo_yolo(char *cfgfile, char *weightfile, float thresh, int cam_index){
    fprintf(stderr, "YOLO demo needs OpenCV for webcam images.\n");
}
#endif
